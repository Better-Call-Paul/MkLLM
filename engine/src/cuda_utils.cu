#include "cuda_utils.cuh"

namespace Engine 
{

void cudaCheck(hipError_t error, const char* file, int line)
{
    if (error != hipSuccess) 
    {
        std::fprintf(stderr,
                     "[CUDA ERROR] %s:%d %s\n",
                     file, line, hipGetErrorString(error));
        std::exit(EXIT_FAILURE);
    }
}

void cudaDeviceInfo()
{
    int deviceId = 0;
    CUDA_CHECK(hipGetDevice(&deviceId));

    hipDeviceProp_t props{};
    CUDA_CHECK(hipGetDeviceProperties(&props, deviceId));

    std::cout
        << "Device ID: "                      << deviceId                                       << "\n"
        << "Name: "                           << props.name                                     << "\n"
        << "Compute Capability: "            << props.major << "." << props.minor               << "\n"
        << "Memory Bus Width: "              << props.memoryBusWidth                            << "\n"
        << "Max Threads Per Block: "         << props.maxThreadsPerBlock                        << "\n"
        << "Max Threads Per Multiprocessor: "<< props.maxThreadsPerMultiProcessor                << "\n"
        << "Regs Per Block: "                << props.regsPerBlock                              << "\n"
        << "Regs Per Multiprocessor: "       << props.regsPerMultiprocessor                     << "\n"
        << "Total Global Memory: "           << (props.totalGlobalMem   / 1024 / 1024) << " MB\n"
        << "Shared Mem Per Block: "          << (props.sharedMemPerBlock / 1024)      << " KB\n"
        << "Shared Mem Per Multiprocessor: " << (props.sharedMemPerMultiprocessor / 1024) << " KB\n"
        << "Total Constant Memory: "         << (props.totalConstMem     / 1024)      << " KB\n"
        << "Multiprocessor Count: "          << props.multiProcessorCount                       << "\n"
        << "Warp Size: "                     << props.warpSize                                  << "\n";
}

}
