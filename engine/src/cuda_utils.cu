#include "cuda_utils.cuh"

namespace Engine 
{

void cudaCheck(hipError_t error, const char* file, int line)
{
    if (error != hipSuccess) 
    {
        std::fprintf(stderr,
                     "[CUDA ERROR] %s:%d %s\n",
                     file, line, hipGetErrorString(error));
        std::exit(EXIT_FAILURE);
    }
}

void cudaDeviceInfo() {
    int id; CUDA_CHECK(hipGetDevice(&id));
    hipDeviceProp_t p; CUDA_CHECK(hipGetDeviceProperties(&p, id));
    std::cout
        << "Device ID: "          << id                    << "\n"
        << "Name: "               << p.name                << "\n"
        << "Compute Capability: " << p.major << "." << p.minor << "\n"
        << "Total Global Mem: "   << (p.totalGlobalMem/1024/1024)
        << " MB\n";
}

}