#include <iostream>
#include <vector>
#include <random>
#include <fstream>
#include <iomanip>
#include <cstdlib>
#include <cmath>
#include <string>
#include <hip/hip_runtime.h>
#include <hipblas.h>

#define CEIL_DIV(x,y) (((x)+(y)-1)/(y))

inline void cudaCheck(hipError_t err, const char* file, int line) {
    if (err != hipSuccess) {
        std::cerr << file << ":" << line
                  << " CUDA Error: " << hipGetErrorString(err)
                  << " (" << err << ")\n";
        std::exit(EXIT_FAILURE);
    }
}
#define CUDA_CHECK(call) cudaCheck(call, __FILE__, __LINE__)

#define CUBLAS_CHECK(call)                                                      \
    do {                                                                         \
        hipblasStatus_t status = (call);                                          \
        if (status != HIPBLAS_STATUS_SUCCESS) {                                   \
            std::cerr << __FILE__ << ":" << __LINE__                             \
                      << " cuBLAS error " << status << "\n";                     \
            std::exit(EXIT_FAILURE);                                             \
        }                                                                        \
    } while(0)

const std::string error_log_file = "matmul_failure_file.txt";

void randomizeMatrix(float* mat, size_t size) {
    static std::mt19937 rng(12345);
    std::uniform_real_distribution<float> dist(0.0f, 1.0f);
    for (size_t i = 0; i < size; ++i) mat[i] = dist(rng);
}

void rangeInitMatrix(float* mat, size_t size) {
    for (size_t i = 0; i < size; ++i) mat[i] = float(i);
}

bool verifyMatrix(const float* ref, const float* out,
                  int M, int N, const std::string& logfile) {
    std::ofstream ofs(logfile, std::ios::app);
    const float tol = 1e-2f;
    bool any_error = false;
    int total = M * N;
    for (int i = 0; i < total; ++i) {
        float d = std::abs(ref[i] - out[i]);
        if (d > tol) {
            if (!any_error) {
                ofs << "=== MISMATCH for " << M << "x" << N << " ===\n";
                any_error = true;
            }
            ofs << "idx " << i
                << ": ref=" << std::setprecision(6) << ref[i]
                << ", out="  << out[i]
                << ", diff=" << d << "\n";
        }
    }
    if (!any_error) {
        ofs << "Verify passed for " << M << "x" << N << "\n";
        return true;
    }
    return false;
}

inline void checkKernelLaunch(int) { }
inline void checkKernelSync(int)  { }

__global__ void sgemm_naive(int M, int N, int K,
                     float alpha, const float *A,
                     const float *B, float beta, float *C) 
{
    int row = blockDim.y * blockIdx.y + threadIdx.y;
    int col = blockDim.x * blockIdx.x + threadIdx.x;

    if (row < M && col < N) 
    {
        float sum = 0.0f;

        for (int i = 0; i < K; ++i) 
        {
            sum += A[row * K + i] * B[i * N + col];
        }
        C[row * N + col] = alpha * sum + C[row * N + col] * beta;
    }

}

template <const uint BLOCKSIZE>
__global__ void sgemm_global_mem_coalesce(
    int M,
    int N,
    int K,
    float alpha,
    const float *A,
    const float *B,
    float beta,
    float *C)
{
    const uint cRow = blockDim.y * BLOCKSIZE + (threadIdx.x / BLOCKSIZE);
    const uint cCol = blockDim.x * BLOCKSIZE + (threadIdx.x % BLOCKSIZE);

    if (cRow < M && cCol < N) 
    {
        float sum = 0.0f;
        for (int i = 0; i < K; ++i) 
        {
            sum += A[cRow * K + i] * B[i * N + cCol];
        }
        C[cRow * N + cCol] = alpha * sum + beta * C[cRow + N + cCol];
    }
}

inline void run_sgemm_coalesced(
    int M,
    int N,
    int K,
    float alpha,
    const float *A,
    const float *B,
    float beta,
    float *C)
{
    constexpr uint blockSize = 32;
    dim3 blockDim(blockSize * blockSize, 1);
    dim3 gridDim(CEIL_DIV(M, blockSize), CEIL_DIV(N, blockSize));
    sgemm_global_mem_coalesce<blockSize><<<gridDim, blockDim>>>(M, N, K, alpha, A, B, beta, C);
}

void run_sgemm_naive(int M, int N, int K,
                     float alpha, const float *A,
                     const float *B, float beta, float *C) {
    dim3 blk(32,32), grid(CEIL_DIV(N,32), CEIL_DIV(M,32));
    sgemm_naive<<<grid,blk>>>(M,N,K,alpha,A,B,beta,C);
}

void run_cublas_sgemm(hipblasHandle_t handle,
                      int M, int N, int K,
                      float alpha,
                      const float *A, const float *B,
                      float beta, float *C) {
    size_t sizeA = sizeof(float) * M * K;
    size_t sizeB = sizeof(float) * K * N;
    size_t sizeC = sizeof(float) * M * N;
    float *dA, *dB, *dC;

    CUDA_CHECK(hipMalloc((void**)&dA, sizeA));
    CUDA_CHECK(hipMalloc((void**)&dB, sizeB));
    CUDA_CHECK(hipMalloc((void**)&dC, sizeC));

    CUDA_CHECK(hipMemcpy(dA, A, sizeA, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(dB, B, sizeB, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(dC, C, sizeC, hipMemcpyHostToDevice));

    CUBLAS_CHECK(hipblasGemmEx(handle,
                              HIPBLAS_OP_N, HIPBLAS_OP_N,
                              N, M, K,
                              &alpha,
                              dB, HIP_R_32F, N,
                              dA, HIP_R_32F, K,
                              &beta,
                              dC, HIP_R_32F, N,
                              HIPBLAS_COMPUTE_32F,
                              CUBLAS_GEMM_DEFAULT_TENSOR_OP));

    CUDA_CHECK(hipDeviceSynchronize());
    CUDA_CHECK(hipMemcpy(C, dC, sizeC, hipMemcpyDeviceToHost));

    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
}

template <const int BLOCKSIZE>
__global__ void sgemm_shmem_cache_blocking(int M, int N, int K, float alpha,
                                       const float *A, const float *B,
                                       float beta, float *C)
{
    const uint cRow = blockIdx.x;
    const uint cCol = blockIdx.y;

    __shared__ float As[BLOCKSIZE * BLOCKSIZE];
    __shared__ float Bs[BLOCKSIZE * BLOCKSIZE];

    const uint threadCol = threadIdx.x % BLOCKSIZE;
    const uint threadRow = threadIdx.x / BLOCKSIZE;

    A += cRow * BLOCKSIZE * K;
    B += cCol * BLOCKSIZE;
    C += cRow * BLOCKSIZE * N + cCol * BLOCKSIZE;

    float tmp = 0.0f;
    for (int bkIdx = 0; bkIdx < K; bkIdx += BLOCKSIZE)
    {
        As[threadRow * BLOCKSIZE + threadCol] = A[threadRow * K + threadCol];
        Bs[threadRow * BLOCKSIZE + threadCol] = B[threadRow * N + threadCol];

        __syncthreads();
        A += BLOCKSIZE;
        B += BLOCKSIZE * N;

        for (int dotIdx = 0; dotIdx < BLOCKSIZE; ++dotIdx)
        {
            tmp += As[threadRow * BLOCKSIZE + dotIdx] *
                   Bs[dotIdx * BLOCKSIZE + threadCol];
        }
        __syncthreads();
    }
    C[threadRow * N + threadCol] = alpha * tmp + beta * C[threadRow * N + threadCol];
}

void run_sgemm_shmem_cache_blocking(int M, int N, int K,
                      float alpha,
                      const float *A, const float *B,
                      float beta, float *C)
{
    constexpr uint block_size = 32;
    dim3 blockDim(block_size * block_size, 1);

    dim3 gridDim(CEIL_DIV(M, block_size), CEIL_DIV(N, block_size));

    hipFuncSetAttribute(reinterpret_cast<const void*>(sgemm_shmem_cache_blocking<32>), hipFuncAttributePreferredSharedMemoryCarveout,
                       cudaSharedmemCarveoutMaxShared);

    sgemm_shmem_cache_blocking<32><<<gridDim, blockDim>>>(M, N, K, alpha, A, B, beta, C);
}

int main()
{
    int deviceIdx = 0;
    if (const char* env = std::getenv("DEVICE"))
    {
        deviceIdx = std::atoi(env);
    }
    CUDA_CHECK(hipSetDevice(deviceIdx));
    std::cout << "Running on device " << deviceIdx << "\n";

    hipblasHandle_t handle;
    CUBLAS_CHECK(hipblasCreate(&handle));

    std::vector<int> sizes = {128, 256, 512, 1024};
    int max_size = sizes.back();
    size_t max_bytes = sizeof(float) * max_size * max_size;

    std::vector<float> A(max_size * max_size);
    std::vector<float> B(max_size * max_size);
    std::vector<float> C(max_size * max_size);
    std::vector<float> C_ref = C;

    randomizeMatrix(A.data(), A.size());
    randomizeMatrix(B.data(), B.size());
    rangeInitMatrix(C.data(), C.size());
    C_ref = C;

    float* dA;
    float* dB;
    float* dC;
    CUDA_CHECK(hipMalloc(&dA, max_bytes));
    CUDA_CHECK(hipMalloc(&dB, max_bytes));
    CUDA_CHECK(hipMalloc(&dC, max_bytes));
    CUDA_CHECK(hipMemcpy(dA, A.data(), max_bytes, hipMemcpyHostToDevice));
    CUDA_CHECK(hipMemcpy(dB, B.data(), max_bytes, hipMemcpyHostToDevice));

    float alpha = 2.0f;
    float beta  = 3.0f;

    {
        int M = sizes[0];
        int N = M;
        int K = M;

        CUDA_CHECK(hipMemcpy(dC, C.data(), sizeof(float) * M * N, hipMemcpyHostToDevice));
        run_sgemm_naive(M, N, K, alpha, dA, dB, beta, dC);
        CUDA_CHECK(hipDeviceSynchronize());
        CUDA_CHECK(hipMemcpy(C.data(), dC, sizeof(float) * M * N, hipMemcpyDeviceToHost));
        run_cublas_sgemm(handle, M, N, K, alpha, A.data(), B.data(), beta, C_ref.data());
        if (!verifyMatrix(C_ref.data(), C.data(), M, N, error_log_file))
        {
            return 1;
        }

        CUDA_CHECK(hipMemcpy(dC, C.data(), sizeof(float) * M * N, hipMemcpyHostToDevice));
        run_sgemm_coalesced(M, N, K, alpha, dA, dB, beta, dC);
        CUDA_CHECK(hipDeviceSynchronize());
        CUDA_CHECK(hipMemcpy(C.data(), dC, sizeof(float) * M * N, hipMemcpyDeviceToHost));
        if (!verifyMatrix(C_ref.data(), C.data(), M, N, error_log_file))
        {
            return 1;
        }

        CUDA_CHECK(hipMemcpy(dC, C.data(), sizeof(float) * M * N, hipMemcpyHostToDevice));
        run_sgemm_shmem_cache_blocking(M, N, K, alpha, dA, dB, beta, dC);
        CUDA_CHECK(hipDeviceSynchronize());
        CUDA_CHECK(hipMemcpy(C.data(), dC, sizeof(float) * M * N, hipMemcpyDeviceToHost));
        if (!verifyMatrix(C_ref.data(), C.data(), M, N, error_log_file))
        {
            std::cout << "Issue with Shmem Cache Blocking Kernel\n";
            return 1;
        }
    }

    const int NUM_RUNS = 10;
    hipEvent_t start;
    hipEvent_t stop;
    CUDA_CHECK(hipEventCreate(&start));
    CUDA_CHECK(hipEventCreate(&stop));

    for (int size : sizes)
    {
        int M = size;
        int N = size;
        int K = size;
        float t_naive   = 0.0f;
        float t_coal    = 0.0f;
        float t_cublas  = 0.0f;
        float t_shared  = 0.0f;

        for (int i = 0; i < NUM_RUNS; ++i)
        {
            CUDA_CHECK(hipMemcpy(dC, C.data(), sizeof(float) * M * N, hipMemcpyHostToDevice));
            CUDA_CHECK(hipEventRecord(start));
            run_sgemm_naive(M, N, K, alpha, dA, dB, beta, dC);
            CUDA_CHECK(hipEventRecord(stop));
            CUDA_CHECK(hipEventSynchronize(stop));
            {
                float ms;
                CUDA_CHECK(hipEventElapsedTime(&ms, start, stop));
                t_naive += ms;
            }

            CUDA_CHECK(hipMemcpy(dC, C.data(), sizeof(float) * M * N, hipMemcpyHostToDevice));
            CUDA_CHECK(hipEventRecord(start));
            run_sgemm_coalesced(M, N, K, alpha, dA, dB, beta, dC);
            CUDA_CHECK(hipEventRecord(stop));
            CUDA_CHECK(hipEventSynchronize(stop));
            {
                float ms;
                CUDA_CHECK(hipEventElapsedTime(&ms, start, stop));
                t_coal += ms;
            }

            CUDA_CHECK(hipMemcpy(dC, C.data(), sizeof(float) * M * N, hipMemcpyHostToDevice));
            CUDA_CHECK(hipEventRecord(start));
            run_cublas_sgemm(handle, M, N, K, alpha, A.data(), B.data(), beta, C_ref.data());
            CUDA_CHECK(hipEventRecord(stop));
            CUDA_CHECK(hipEventSynchronize(stop));
            {
                float ms;
                CUDA_CHECK(hipEventElapsedTime(&ms, start, stop));
                t_cublas += ms;
            }

            CUDA_CHECK(hipMemcpy(dC, C.data(), sizeof(float) * M * N, hipMemcpyHostToDevice));
            CUDA_CHECK(hipEventRecord(start));
            run_sgemm_shmem_cache_blocking(M, N, K, alpha, dA, dB, beta, dC);
            CUDA_CHECK(hipEventRecord(stop));
            CUDA_CHECK(hipEventSynchronize(stop));
            {
                float ms;
                CUDA_CHECK(hipEventElapsedTime(&ms, start, stop));
                t_shared += ms;
            }
        }

        std::cout
            << "dim " << size
            << " | naive: "   << (t_naive   / NUM_RUNS) << " ms"
            << " | coal: "    << (t_coal    / NUM_RUNS) << " ms"
            << " | cuBLAS: "  << (t_cublas  / NUM_RUNS) << " ms"
            << " | shared: "  << (t_shared  / NUM_RUNS) << " ms\n";
    }

    CUDA_CHECK(hipFree(dA));
    CUDA_CHECK(hipFree(dB));
    CUDA_CHECK(hipFree(dC));
    CUDA_CHECK(hipEventDestroy(start));
    CUDA_CHECK(hipEventDestroy(stop));
    CUBLAS_CHECK(hipblasDestroy(handle));

    return 0;
}
